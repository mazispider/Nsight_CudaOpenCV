#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<hip/hip_runtime.h>

using std::cout;
using std::endl;

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg, file_name, line_number, hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)
#define DIM 8192

struct hipComplex {
	float   r;
	float   i;
	// hipComplex( float a, float b ) : r(a), i(b)  {}
	__device__ hipComplex(float a, float b) : r(a), i(b) {} // Fix error for calling host function from device
	__device__ float magnitude2(void) {
		return r * r + i * i;
	}
	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i * a.i, i*a.r + r * a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r + a.r, i + a.i);
	}
};

__device__ int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for (i = 0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

__global__ void kernel(unsigned char *ptr) {
	// map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y * DIM;

	// now calculate the value at that position
	if (x < DIM || y < DIM) {
		int juliaValue = julia(x, y);
		ptr[offset*3] = 0;
		ptr[offset*3 + 1] = 0;
		ptr[offset*3 + 2] = 255 * juliaValue;
	}
}

void julia(cv::Mat& output)
{
	//Calculate total number of bytes of input and output image

	const int grayBytes = output.step * output.rows;

	unsigned char  *d_output;
	std::cout << grayBytes << std::endl;
	//Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");

	//Copy data from OpenCV input image to device memory


	//Specify a reasonable block size
	const dim3 block(16, 16);

	//Calculate grid size to cover the whole image
	const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

	//Launch the color conversion kernel
	kernel <<<grid, block >>>(d_output);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	//Create output image
	cv::Mat output(DIM, DIM, CV_8UC3);
	std::cout << output.cols << std::endl;
	hipEventRecord(start,0);
	//Call the wrapper function
	julia(output);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float   elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "time to generate " << 1000 * elapsedTime << "us." << std::endl;
	//Show the input and output
	//cv::imshow("Input", input);
	//cv::imshow("Output", output);
	cv::imwrite("./julia.jpg", output);
	//Wait for key press
	cv::waitKey();
	std::cin.get();
	return 0;
}
